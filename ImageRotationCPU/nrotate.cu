#include "hip/hip_runtime.h"
#include "ImageStuff.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""

struct ConstantValues
{
	double diagonal;
	double scaleFactor;
	double cosineScaled;
	double sineScaled;
	int xCenterOfImage;
	int yCenterOfImage;
}; // end of struct constantValues

void readCommandline( unsigned char ** originalImage, int * numberOfFrames, char * outputFilename, int argc, char ** argv )
{
	if( argc != 4 )
	{
		printf( "Usage: %s <input image> <output image> <number of rotations>\n", argv[0] );
		exit( EXIT_FAILURE );
	}

	*originalImage = readBMP( argv[1] );
	strcpy( outputFilename, argv[2] );
	*numberOfFrames = atoi( argv[3] );

	if( *numberOfFrames > 30 )
	{
		printf( "Number of frame should not exceed 30.\n" );
		exit( EXIT_FAILURE );
	}

	// Discard file type extension
	int i;
	for( i = 0; outputFilename[i] != '.'; i++ )
		; // empty loop

	outputFilename[i] = 0;

} // end of function readCommandline

void cleaning( unsigned char ** originalImage )
{
	free( *originalImage );
} // end of function cleaning

void checkCudaStatus( hipError_t status )
{
	if( status == hipSuccess )
		return;

	printf( "Error occurred in CUDA API" );
	exit( EXIT_FAILURE );
} // end of function checkCudaStatus

__global__ void rotateKernel( unsigned char * rotatedImage, const unsigned char * inputImage, const double angle,
		const int numberOfRows, const int numberOfCols, struct ConstantValues preCalculatedValues)
{
	int row;
	int col;
	double pixelCartesianX;
	double pixelCartesianY;
	double sinPixelY;
	double cosPixelY;
	double newX;
	double newY;
	int newRow;
	int newCol;

	row = threadIdx.y + blockIdx.y * blockDim.y;
	col = threadIdx.x + blockIdx.x * blockDim.x;


	// Out of picture!
	if( row > numberOfRows || col*3 > numberOfCols )
		return;

	pixelCartesianY = preCalculatedValues.yCenterOfImage - ( double ) row;
	sinPixelY = pixelCartesianY * preCalculatedValues.sineScaled;
	cosPixelY = pixelCartesianY * preCalculatedValues.cosineScaled;
	pixelCartesianX = ( double ) ( col - preCalculatedValues.xCenterOfImage );
	newX = preCalculatedValues.cosineScaled * pixelCartesianX - sinPixelY;
	newY = preCalculatedValues.sineScaled * pixelCartesianX + cosPixelY;
	newRow = preCalculatedValues.yCenterOfImage - ( int ) newY;
	newCol = ( double ) ( preCalculatedValues.xCenterOfImage + newX );
	col *= 3;

	if( newCol >= 0 && newRow >= 0 && newCol < numberOfCols && newRow < numberOfRows )
	{
		newCol *= 3;
		rotatedImage[newRow * numberOfCols + newCol] = inputImage[row * numberOfCols + col];
		rotatedImage[newRow * numberOfCols + newCol + 1] = inputImage[row * numberOfCols + col + 1];
		rotatedImage[newRow * numberOfCols + newCol + 2] = inputImage[row * numberOfCols + col + 2];
	}

	//rotatedImage[row * numberOfCols + col ] = inputImage[row * numberOfCols + col];
	//rotatedImage[row * numberOfCols + col + 1 ] = inputImage[row * numberOfCols + col + 1];
	//rotatedImage[row * numberOfCols + col + 2 ] = inputImage[row * numberOfCols + col + 2];
} // end of kernel rotateKernel

void calculateConstantValues( struct  ConstantValues * constantValues, const double rotateAngle )
{
	const int numberOfRows = ip.Vpixels;
	const int numberOfCols = ip.Hpixels;

	constantValues->diagonal = sqrt ( ( double ) ( numberOfRows * numberOfRows + numberOfCols * numberOfCols ) );
	constantValues->scaleFactor = ( numberOfCols > numberOfRows ) ? ( ( double ) numberOfRows ) / constantValues->diagonal : ( ( double ) numberOfCols ) / constantValues->diagonal;
	constantValues->cosineScaled = cos( rotateAngle ) * constantValues->scaleFactor;
	constantValues->sineScaled = sin( rotateAngle ) * constantValues->scaleFactor;
	constantValues->xCenterOfImage = numberOfCols / 2;
	constantValues->yCenterOfImage = numberOfRows / 2;

} // end of function calculateConstantValues

void launchGPU( const unsigned char * originalImage, unsigned char * resultImage, const double angle, const int numberOfFrames,
		const char * outputFilename, const int i )
{
	const int maximumStringSize = 50;
	const int blockSize = 32;
	const int height = ip.Vpixels;
	const int width = ip.Hbytes;
	const int backgroundColor = 0;

	hipError_t status;
	unsigned char * gpuInput;
	unsigned char * gpuResult;
	int gpuArraySize;
	dim3 grid;
	dim3 block;
	char filename[maximumStringSize];
	char appendix[2];
	double currentAngle;
	struct ConstantValues preCalculatedVals;

	block = dim3( blockSize, blockSize );
	grid = dim3( ( width / blockSize ) + 1, ( height / blockSize ) + 1 );
	gpuArraySize = height * width * sizeof( unsigned char );
	currentAngle = i * angle;

	status = hipSetDevice( 0 );
	checkCudaStatus( status );

	status = hipMalloc( ( void ** ) &gpuInput, gpuArraySize );
	checkCudaStatus( status );

	status = hipMalloc( ( void ** ) &gpuResult, gpuArraySize );
	checkCudaStatus( status );

	status = hipMemcpy( gpuInput, originalImage, gpuArraySize, hipMemcpyHostToDevice );
	checkCudaStatus( status );

	status = hipMemset( gpuResult, backgroundColor, gpuArraySize );
	checkCudaStatus( status );

	calculateConstantValues( &preCalculatedVals, currentAngle );

	rotateKernel<<< grid, block >>>( gpuResult, gpuInput, currentAngle, height, width, preCalculatedVals );
	hipDeviceSynchronize();

	status = hipMemcpy( resultImage, gpuResult, gpuArraySize, hipMemcpyDeviceToHost );
	checkCudaStatus( status );

	strcpy( filename, outputFilename );
	sprintf( appendix, "%02d", i);
	strcat( filename, appendix );
	strcat( filename, ".bmp" );

	writeBMP( resultImage, filename );

	hipFree( gpuInput );
	hipFree( gpuResult );

} // end of function launchGPU

void rotateNTimes( const unsigned char * originalImage, const int numberOfFrames, const char * outputFilename )
{
	int rotationDegree;
	double rotationAngleRadian;
	const double pi = 3.141592;
	unsigned char * rotatedImaage;

	rotationDegree = 360 / numberOfFrames;
	rotationAngleRadian =  pi / 180.0  * ( double ) rotationDegree;

	rotatedImaage = ( unsigned char * ) malloc( ip.Hbytes * ip.Vpixels * sizeof( unsigned char ) );

	if( rotatedImaage == NULL )
	{
		printf( "Failed to allocated memory for rotated image.\n" );
		exit( EXIT_FAILURE );
	}

	for( int i = 0; i < numberOfFrames; i++ )
		launchGPU( originalImage, rotatedImaage, rotationAngleRadian, numberOfFrames, outputFilename, i );

} // end of function rotateNTimes


int main( int argc, char ** argv )
{
	const int stringSizeMax = 50;
	unsigned char * originalImage;
	int numberOfFrames;
	char outputFilename[stringSizeMax];

	readCommandline( &originalImage, &numberOfFrames, outputFilename, argc, argv );
	rotateNTimes( originalImage, numberOfFrames, outputFilename );

	cleaning( &originalImage );
	printf( "Done!" );
	return 0;
} // end of function main
